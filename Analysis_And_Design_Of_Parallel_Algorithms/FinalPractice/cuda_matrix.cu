
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

typedef int* Vector;

int techo(int tam, int block_size)
{
	return (tam / block_size) + (tam % block_size == 0? 0: 1);
}
__global__ void multiplicarMatrices(int *C, int *A, int *B, int tam)
{

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int index = idy * tam + idx;
	
	if(idy < tam && idx < tam)
	{
		int suma = 0;
		for(int k = 0; k < tam; k++)
			suma += A[idy + tam + k] * B [k * tam + idx];
		C[index] = suma;
	}
}

__global__ void sumarMatrices(int *C, int *A, int *B, int tam)
{

        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        int idy = blockIdx.y * blockDim.y + threadIdx.y;
        int index = idy * tam + idx;

        if(idy < tam && idx < tam)
            C[index] = A[index] + B[index];
}


void imprimirMatriz(int *A, int tam)
{
	for(int i = 0; i < tam; i++)
	{
		for(int j = 0; j < tam; j++)
		{
			printf("%d ", A[tam * i + j]);
		}
		printf("\n");
	}
}
Vector newMatrizVectorizada(int tam, int val)
{
	Vector v = (Vector) malloc(sizeof(int) * tam * tam);
	for(int i = 0; i < tam; i++)
	{
		for(int j = 0; j < tam; j++)
		{
			v[i * tam + j] = val;
		}
	}
	return v;
}
int main(int argc, char **argv)
{
	Vector A_h, B_h, C_h;
	Vector A_d, B_d, C_d;
	
	int tam = 12;
	if(argc > 1)
		tam = atoi(argv[1]);
	int N = tam * tam;
	
	size_t  size = N * sizeof(int);
	
	A_h = newMatrizVectorizada(tam, 2);
	B_h = newMatrizVectorizada(tam, 3);
	C_h = newMatrizVectorizada(tam, 0);
	
	hipMalloc((void**) &A_d, size);
	hipMalloc((void**) &B_d, size);
	hipMalloc((void**) &C_d, size);
	
	hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
	hipMemcpy(B_d,	B_h, size, hipMemcpyHostToDevice);
	
	int BLOCK_SIZE = 4;
	dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
	dim3 n_blocks(techo(tam, block_size.x), techo(tam, block_size.y));
	
	multiplicarMatrices<<< n_blocks, block_size >>> (C_d, A_d, B_d, tam);
	
	hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);

	imprimirMatriz(C_h, tam); 

	sumarMatrices <<< n_blocks, block_size>>> (C_d, A_d, B_d, tam);
	hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);
	printf("==============================================RESULTADO============================================\n");
	imprimirMatriz(C_h, tam);
	printf("===================================================================================================\n");
	
	free(A_h);
	free(B_h);
	free(C_h);

	hipFree(A_d);
	hipFree(B_d);
	hipFree(C_d);

	return 0;
	
}
